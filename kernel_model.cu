#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <malloc.h>
#include <stdio.h>
#include <math.h>
#include <time.h>

#define N XYXY //Tamanho da Malha
#define dom 50.0
#define IT 300 //Loops temporais

// Sine-Gordon without borders Kernels
__global__ void sineGordon_Kernel(double *m_s,double *m_act, double *m_previous,double dts, double dxts) {
	int i = blockIdx.x;
	int j = threadIdx.x;
	if( (i > 0 && i < N-1) && (j > 0 && j < N-1 ))
	m_act[i*blockDim.x + j] = -m_previous[i*blockDim.x +j] + 2.0 * (1.0 - 2.0 * dxts) * m_s[i*blockDim.x + j] + dxts * (m_s[(i + 1)*blockDim.x+j] + m_s[(i - 1)*blockDim.x+j] + m_s[i*blockDim.x+j + 1] + m_s[i*blockDim.x+j - 1]) - dts * (sin((m_s[(i + 1)*blockDim.x+j] + m_s[(i - 1)*blockDim.x+j] + m_s[i*blockDim.x+j + 1] + m_s[i*blockDim.x+j - 1]) / 4.0));
}

__global__ void actualization_Kernel(double* m_s, double* m_act, double* m_previous) {
	int i = blockIdx.x;
	int j = threadIdx.x;
	if ((i > 0 && i < N-1) && (j > 0 && j < N-1)) {
		m_previous[i * blockDim.x + j] = m_s[i * blockDim.x + j];
		m_s[i * blockDim.x + j] = m_act[i * blockDim.x + j];
	}
}

void showMatrix(double *m) {
	int i, j;

	for ( i = 0; i < N; i++)
	{
		for ( j = 0; j < N; j++)
		{
			printf("%lf\t",m[i*N + j]);
		}
		printf("\n");
	}
}

void inicMatrix(double *m) {
	int i, j;

	for (i = 0; i < N; i++)
	{
		for (j = 0; j < N; j++)
		{
			m[i * N + j] = 0;
		}
	}
}

void initialCond(double* m, double dx, double dy) {
	int i, j;
	double x, y;
	y = -dom/2.0;
	for (j = 0; j < N; j++) {
		x = -dom/2.0;
		for (i = 0; i < N; i++) {
			m[i*N+j] = 4 * atan(exp(3 - sqrt(x * x + y * y)));
			x = x + dx;
		}
		y = y + dy;
	}

}

void deviceCapabilities() {

	hipDeviceProp_t prop;
	int count;
	hipGetDeviceCount(&count);
	for (int i = 0; i < count; i++) {
		hipGetDeviceProperties(&prop, i);
		printf(" --- General Information for device %d ---\n", i);
		printf("Name: %s\n", prop.name);
		printf("Compute capability: %d.%d\n", prop.major, prop.minor);
		printf("Clock rate: %d\n", prop.clockRate);
		printf("Device copy overlap: ");
		if (prop.deviceOverlap)
			printf("Enabled\n");
		else
			printf("Disabled\n");
		printf("Kernel execition timeout : ");
		if (prop.kernelExecTimeoutEnabled)
			printf("Enabled\n");
		else
			printf("Disabled\n");
		printf(" --- Memory Information for device %d ---\n", i);
		printf("Total global mem: %ld\n", prop.totalGlobalMem);
		printf("Total constant Mem: %ld\n", prop.totalConstMem);
		printf("Max mem pitch: %ld\n", prop.memPitch);
		printf("Texture Alignment: %ld\n", prop.textureAlignment);
		printf(" --- MP Information for device %d ---\n", i);
		printf("Multiprocessor count: %d\n",
			prop.multiProcessorCount);
		printf("Shared mem per mp: %ld\n", prop.sharedMemPerBlock);
		printf("Registers per mp: %d\n", prop.regsPerBlock);
		printf("Threads in warp: %d\n", prop.warpSize);
		printf("Max threads per block: %d\n",
			prop.maxThreadsPerBlock);
		printf("Max thread dimensions: (%d, %d, %d)\n",
			prop.maxThreadsDim[0], prop.maxThreadsDim[1],
			prop.maxThreadsDim[2]);
		printf("Max grid dimensions: (%d, %d, %d)\n",
			prop.maxGridSize[0], prop.maxGridSize[1],
			prop.maxGridSize[2]);
		printf("\n");
	}
}

int  main() {
	double *hm_s, *hm_act,*hm_previous, dx, dy, dt, *dxts, *dts; //Host variables
	double *dm_s, *dm_act, *dm_previous, *d_dxts, *d_dts; // Device Variables
	int i, j, k;

	dts = (double*)malloc(sizeof(double));
	dxts = (double*)malloc(sizeof(double));

	//Defini��o de Par�metros
	printf("Definindo parametros para a discretizao ... \n");
	dx = dom / N;
	dy = dom / N;
	dt = dx / sqrt(2.0);
	*dts = dt * dt;
	*dxts = (dt / dx) * (dt / dx);

	//deviceCapabilities();

	//printf("Alocando mem�ria no HOST ... \n");
	hm_s = (double*)malloc((N * N) * sizeof(double)); // Matrix Solution on HOST
	hm_act = (double*)malloc((N * N) * sizeof(double)); // Matrix Actualizations on HOST 
	hm_previous = (double*)malloc((N * N) * sizeof(double)); //Previous Matrix results on HOST

	printf("Inicializando matrizes no HOST ... \n");
	inicMatrix(hm_s); // Zeros  Matrix
	inicMatrix(hm_act); // Zeros  Matrix
	inicMatrix(hm_previous); // Zeros  Matrix 

	printf("Aplicando as condicoes iniciais a matriz ... \n");
	initialCond(hm_s, dx, dy); // Appling initial conditions


	printf("Alocando memoria no DEVICE ... \n");
	hipMalloc(&dm_s, (N * N) * sizeof(double)); // Matrix Solution on Device
	hipMalloc(&dm_act, (N * N) * sizeof(double)); // Matrix Actualizations on Device
	hipMalloc(&dm_previous, (N * N) * sizeof(double)); // Matrix Actualizations on Device
	hipMalloc(&d_dts, sizeof(double)); // dts on Device
	hipMalloc(&d_dxts, sizeof(double)); // dxts on Device

	//deviceCapabilities();

	hipMemcpy(d_dts, dts, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_dxts, dxts, sizeof(double), hipMemcpyHostToDevice);
	
	clock_t begin = clock();
	printf("Temporal Evolution ... \n");
	for ( i = 0; i < IT; i++)
	{
		//printf("Transferindo informacoes do HOST para o DEVICE ... \n");
		hipMemcpy(dm_s, hm_s, (N * N) * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(dm_act, hm_act, (N * N) * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(dm_previous, hm_previous, (N * N) * sizeof(double), hipMemcpyHostToDevice);

		// Parellelism in N blocks and N threads per block for the inner elements 
		sineGordon_Kernel <<< N, N >>> (dm_s, dm_act, dm_previous, *dts, *dxts); // Parallelism in N Blocks with N Threads
		//actualization_Kernel << < N, N >> > (dm_s, dm_act, dm_previous);

		//printf("Transferindo atualizacoes do DEVICE para o HOST ... \n");
		hipMemcpy(hm_act, dm_act, (N * N) * sizeof(double), hipMemcpyDeviceToHost);
		
		for (k = 1; k < N - 1; k++) {
			for ( j = 1; j < N - 1 ; j++)
			{
				hm_previous[k * N + j] = hm_s[k * N + j];
				hm_s[k * N + j] = hm_act[k * N + j];
			}
		}
	}

	clock_t end = clock();
	double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
	FILE *pf;
	pf = fopen("tempo.dat","a");
	fprintf(pf,"%lf\n",time_spent);
	
	printf("Liberando memoria no HOST e no DEVICE ... \n");
	// Unallocing CPU variables
	free(hm_s);
	free(hm_act);
	free(hm_previous);
	free(dxts);
	free(dts);
	//Unallocing GPU variables
	hipFree(dm_s); 
	hipFree(dm_act);
	hipFree(dm_previous);
	hipFree(d_dxts);
	hipFree(d_dts);
	printf("Tempo de Execucao : %lf\n", time_spent);
	return 0;	
}
